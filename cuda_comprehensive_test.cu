#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define THREADS_PER_BLOCK 256

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

// Function to verify the result
void verify_result(float *A, float *B, float *C, int numElements)
{
    for (int i = 0; i < numElements; i++)
    {
        float sum = A[i] + B[i];
        if (fabs(C[i] - sum) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Results verified successfully.\n");
}

int main(void)
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    verify_result(h_A, h_B, h_C, N);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Print CUDA device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Total Global Memory: %zu bytes\n", prop.totalGlobalMem);
    printf("Shared Memory per Block: %zu bytes\n", prop.sharedMemPerBlock);
    printf("Registers per Block: %d\n", prop.regsPerBlock);
    printf("Warp Size: %d\n", prop.warpSize);
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads Dimension: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max Grid Size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    printf("Test PASSED\n");

    return 0;
}
